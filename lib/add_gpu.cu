#include "hip/hip_runtime.h"
#include "add_gpu.cuh"

__global__ void vecAdd(float *a, float *b, float *c, int n)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n)
    {
        c[idx] = a[idx] + b[idx];
    }
}

void add_gpu(float *a, float *b, float *c, int n)
{
    float *d_a;
    float *d_b;
    float *d_c;

    hipMalloc(&d_a, n * sizeof(float));
    hipMalloc(&d_b, n * sizeof(float));
    hipMalloc(&d_c, n * sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n * sizeof(float), hipMemcpyHostToDevice);
    
    int blockSize = 1024;
    int gridSize = (int) ceil((float) n / blockSize);

    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);

    hipMemcpy(a, d_a, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, n * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, n * sizeof(float), hipMemcpyDeviceToHost);
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
